#include "hip/hip_runtime.h"

#include "test.h"

__global__ void record_thread(float *x) {
    int idx = threadIdx.x;
    x[idx] = idx;
}

void kernel(float* data,int N) {
    record_thread<<<1, N>>>(data);
}

__global__ void jsum(float *a, float *b, float *c, float *d, int N) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    while(idx<N) {
        d[idx] = a[idx]*b[idx]+c[idx];
        idx += num_threads;
    }
}

void jsum_host(float *a, float* b,float *c,float*d, int N) {
    jsum<<<(N/256)+1, 256>>>(a,b,c,d,N);
}
